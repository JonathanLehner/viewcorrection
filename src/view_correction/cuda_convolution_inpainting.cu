#include "hip/hip_runtime.h"
// Copyright 2018 ETH Zürich
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include "view_correction/cuda_convolution_inpainting.cuh"

#include <hipcub/hipcub.hpp>
#include <glog/logging.h>

#include "view_correction/cuda_util.h"

namespace view_correction {

constexpr int kIterationsPerKernelCall = 4;

const int kBlockWidth = 32;
const int kBlockHeight = 32;

constexpr float kSqrt2 = 1.4142135623731f;

template<int block_size_x, int block_size_y>
__global__ void ConvolutionInpaintingInitializeVariablesKernel(
    int grid_dim_x,
    float depth_input_scaling_factor,
    hipTextureObject_t depth_map_input,
    CUDABuffer_<float> depth_map_output,
    CUDABuffer_<uint16_t> block_coordinates) {
  const int width = depth_map_output.width();
  const int height = depth_map_output.height();
  
  const int kBlockOutputSizeX = block_size_x - 2 * kIterationsPerKernelCall;
  const int kBlockOutputSizeY = block_size_y - 2 * kIterationsPerKernelCall;
  unsigned int x = blockIdx.x * kBlockOutputSizeX + threadIdx.x - kIterationsPerKernelCall;
  unsigned int y = blockIdx.y * kBlockOutputSizeY + threadIdx.y - kIterationsPerKernelCall;
  
  const bool kOutput =
      threadIdx.x >= kIterationsPerKernelCall &&
      threadIdx.y >= kIterationsPerKernelCall &&
      threadIdx.x < block_size_x - kIterationsPerKernelCall &&
      threadIdx.y < block_size_y - kIterationsPerKernelCall &&
      x < width &&
      y < height;
  
  bool thread_is_active = false;
  if (kOutput) {
    const float depth_input = depth_input_scaling_factor * tex2D<float>(depth_map_input, x, y);
    depth_map_output(y, x) = depth_input;
    thread_is_active = (depth_input == 0);
  }
  
  typedef hipcub::BlockReduce<
      int, block_size_x, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, block_size_y> BlockReduceInt;
  __shared__ typename BlockReduceInt::TempStorage int_storage;
  int num_active_threads = BlockReduceInt(int_storage).Sum(thread_is_active ? 1 : 0);
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    block_coordinates(0, blockIdx.x + blockIdx.y * grid_dim_x) = num_active_threads;
  }
}

template<int block_size_x, int block_size_y, bool check_convergence>
__global__ void ConvolutionInpaintingKernel(
    CUDABuffer_<uint16_t> block_coordinates,
    hipTextureObject_t depth_map_input,
    CUDABuffer_<uint8_t> max_change,
    float max_change_rate_threshold,
    CUDABuffer_<float> depth_map_output) {
  const int x = max(0, min(depth_map_output.width() - 1, block_coordinates(0, 2 * blockIdx.x + 0) + threadIdx.x - kIterationsPerKernelCall));
  const int y = max(0, min(depth_map_output.height() - 1, block_coordinates(0, 2 * blockIdx.x + 1) + threadIdx.y - kIterationsPerKernelCall));
  
  const bool kIsPixelToInpaint = (tex2D<float>(depth_map_input, x, y) <= 0);
  const bool kOutput =
      threadIdx.x >= kIterationsPerKernelCall &&
      threadIdx.y >= kIterationsPerKernelCall &&
      threadIdx.x < block_size_x - kIterationsPerKernelCall &&
      threadIdx.y < block_size_y - kIterationsPerKernelCall &&
      block_coordinates(0, 2 * blockIdx.x + 0) + threadIdx.x - kIterationsPerKernelCall < depth_map_output.width() &&
      block_coordinates(0, 2 * blockIdx.x + 1) + threadIdx.y - kIterationsPerKernelCall < depth_map_output.height();
  
  // Load inputs into private or shared memory.
  __shared__ float depth_shared[block_size_x * block_size_y];
  int shared_mem_index = threadIdx.x + block_size_x * threadIdx.y;
  depth_shared[shared_mem_index] = depth_map_output(y, x);
  
  // Wait for shared memory to be loaded.
  __syncthreads();
  
#pragma unroll
  for (int i = 0; i < kIterationsPerKernelCall; ++ i) {
    float result = 0;
    float weight = 0;
    float pixel_weight;
    float temp_depth;
    if (kIsPixelToInpaint &&
        threadIdx.x > 0 &&
        threadIdx.y > 0 &&
        threadIdx.x < block_size_x - 1 &&
        threadIdx.y < block_size_y - 1) {
      temp_depth = depth_shared[shared_mem_index - 1 - block_size_x];
      pixel_weight =
          (y > 0 && x > 0 && temp_depth > 0) *
          0.073235f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index - block_size_x];
      pixel_weight =
          (y > 0 && temp_depth > 0) *
          0.176765f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index + 1 - block_size_x];
      pixel_weight =
          (y > 0 && x < depth_map_output.width() - 1 && temp_depth > 0) *
          0.073235f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index - 1];
      pixel_weight =
          (x > 0 && temp_depth > 0) *
          0.176765f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index + 1];
      pixel_weight =
          (x < depth_map_output.width() - 1 && temp_depth > 0) *
          0.176765f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index - 1 + block_size_x];
      pixel_weight =
          (y < depth_map_output.height() - 1 && x > 0 && temp_depth > 0) *
          0.073235f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index + block_size_x];
      pixel_weight =
          (y < depth_map_output.height() - 1 && temp_depth > 0) *
          0.176765f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      temp_depth = depth_shared[shared_mem_index + 1 + block_size_x];
      pixel_weight =
          (y < depth_map_output.height() - 1 && x < depth_map_output.width() - 1 && temp_depth > 0) *
          0.073235f;
      result += pixel_weight * temp_depth;
      weight += pixel_weight;
      
      // Version without explicit handling of uninitialized values:
//       result = 0.073235f * depth_shared[shared_mem_index - 1 - block_size_x] +
//                0.176765f * depth_shared[shared_mem_index - block_size_x] +
//                0.073235f * depth_shared[shared_mem_index + 1 - block_size_x] +
//                0.176765f * depth_shared[shared_mem_index - 1] +
//                0 +
//                0.176765f * depth_shared[shared_mem_index + 1] +
//                0.073235f * depth_shared[shared_mem_index - 1 + block_size_x] +
//                0.176765f * depth_shared[shared_mem_index + block_size_x] +
//                0.073235f * depth_shared[shared_mem_index + 1 + block_size_x];
    }
    __syncthreads();
    
    float new_depth = result / weight;
    
    // Convergence test.
    float change = 0;
    if (check_convergence && kOutput && kIsPixelToInpaint && i == kIterationsPerKernelCall - 1) {
      change = fabs((new_depth - depth_shared[shared_mem_index]) / depth_shared[shared_mem_index]);
    }
    if (check_convergence) {
      typedef hipcub::BlockReduce<
          int, block_size_x, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, block_size_y> BlockReduceInt;
      __shared__ typename BlockReduceInt::TempStorage int_storage;
      int active_pixels = BlockReduceInt(int_storage).Sum(change > max_change_rate_threshold);
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        max_change(0, blockIdx.x) = (active_pixels > 0) ? 1 : 0;
      }
    }
    
    if (kIsPixelToInpaint && weight > 0) {
      depth_shared[shared_mem_index] = new_depth;
    }
    __syncthreads();
  }
  
  if (kOutput && kIsPixelToInpaint) {
    depth_map_output(y, x) = depth_shared[shared_mem_index];
  }
}

template<int block_size_x, int block_size_y, bool check_convergence>
__global__ void ConvolutionInpaintingKernelWithWeighting(
    CUDABuffer_<uint16_t> block_coordinates,
    hipTextureObject_t depth_map_input,
    hipTextureObject_t gradient_magnitude_div_sqrt2,
    CUDABuffer_<uint8_t> max_change,
    float max_change_rate_threshold,
    CUDABuffer_<float> depth_map_output) {
  const int raw_x = block_coordinates(0, 2 * blockIdx.x + 0) + threadIdx.x - kIterationsPerKernelCall;
  const int raw_y = block_coordinates(0, 2 * blockIdx.x + 1) + threadIdx.y - kIterationsPerKernelCall;
  const bool kInImage =
      raw_x >= 0 &&
      raw_y >= 0 &&
      raw_x < depth_map_output.width() &&
      raw_y < depth_map_output.height();
  const int x = max(0, min(depth_map_output.width() - 1, raw_x));
  const int y = max(0, min(depth_map_output.height() - 1, raw_y));
  
  const bool kIsPixelToInpaint = (tex2D<float>(depth_map_input, x, y) <= 0);
  const bool kOutput =
      threadIdx.x >= kIterationsPerKernelCall &&
      threadIdx.y >= kIterationsPerKernelCall &&
      threadIdx.x < block_size_x - kIterationsPerKernelCall &&
      threadIdx.y < block_size_y - kIterationsPerKernelCall &&
      kInImage && kIsPixelToInpaint;
  
  // Load inputs into private or shared memory.
  __shared__ float depth_shared[block_size_x * block_size_y];
  __shared__ float weights_shared[block_size_x * block_size_y];
  const int shared_mem_index = threadIdx.x + block_size_x * threadIdx.y;
  depth_shared[shared_mem_index] = depth_map_output(y, x);
  const float base_weight = (kInImage ? 1 : 0) *  1.f / (1.f + 50.f * tex2D<uchar>(gradient_magnitude_div_sqrt2, x, y) * kSqrt2 / 255.f);
  weights_shared[shared_mem_index] = base_weight * (depth_shared[shared_mem_index] > 0);
  
  // Wait for shared memory to be loaded.
  __syncthreads();
  
#pragma unroll
  for (int i = 0; i < kIterationsPerKernelCall; ++ i) {
    float new_depth = 0;
    if (kIsPixelToInpaint &&
        threadIdx.x > 0 &&
        threadIdx.y > 0 &&
        threadIdx.x < block_size_x - 1 &&
        threadIdx.y < block_size_y - 1) {
      float weight = 0;
      float pixel_weight;
      
      pixel_weight =
          0.073235f * weights_shared[shared_mem_index - 1 - block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index - 1 - block_size_x];
      weight += pixel_weight;
      
      pixel_weight =
          0.176765f * weights_shared[shared_mem_index - block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index - block_size_x];
      weight += pixel_weight;
      
      pixel_weight =
          0.073235f * weights_shared[shared_mem_index + 1 - block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index + 1 - block_size_x];
      weight += pixel_weight;
      
      pixel_weight =
          0.176765f * weights_shared[shared_mem_index - 1];
      new_depth += pixel_weight * depth_shared[shared_mem_index - 1];
      weight += pixel_weight;
      
      pixel_weight =
          0.176765f * weights_shared[shared_mem_index + 1];
      new_depth += pixel_weight * depth_shared[shared_mem_index + 1];
      weight += pixel_weight;
      
      pixel_weight =
          0.073235f * weights_shared[shared_mem_index - 1 + block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index - 1 + block_size_x];
      weight += pixel_weight;
      
      pixel_weight =
          0.176765f * weights_shared[shared_mem_index + block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index + block_size_x];
      weight += pixel_weight;
      
      pixel_weight =
          0.073235f * weights_shared[shared_mem_index + 1 + block_size_x];
      new_depth += pixel_weight * depth_shared[shared_mem_index + 1 + block_size_x];
      weight += pixel_weight;
      
      // Version without explicit handling of uninitialized values:
      // (And without weights):
//       result = 0.073235f * depth_shared[shared_mem_index - 1 - block_size_x] +
//                0.176765f * depth_shared[shared_mem_index - block_size_x] +
//                0.073235f * depth_shared[shared_mem_index + 1 - block_size_x] +
//                0.176765f * depth_shared[shared_mem_index - 1] +
//                0 +
//                0.176765f * depth_shared[shared_mem_index + 1] +
//                0.073235f * depth_shared[shared_mem_index - 1 + block_size_x] +
//                0.176765f * depth_shared[shared_mem_index + block_size_x] +
//                0.073235f * depth_shared[shared_mem_index + 1 + block_size_x];
      
      new_depth = new_depth / weight;
    }
    __syncthreads();
    
    // Convergence test.
    if (check_convergence && i == kIterationsPerKernelCall - 1) {
      float change = 0;
      if (kOutput) {
        change = fabs((new_depth - depth_shared[shared_mem_index]) / depth_shared[shared_mem_index]);
      }
      
      typedef hipcub::BlockReduce<
          int, block_size_x, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, block_size_y> BlockReduceInt;
      __shared__ typename BlockReduceInt::TempStorage int_storage;
      int active_pixels = BlockReduceInt(int_storage).Sum(change > max_change_rate_threshold);
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        max_change(0, blockIdx.x) = (active_pixels > 0) ? 1 : 0;
      }
    }
    
    if (kIsPixelToInpaint && new_depth > 0) {
      depth_shared[shared_mem_index] = new_depth;
      if (i < kIterationsPerKernelCall - 1) {
        weights_shared[shared_mem_index] = base_weight * (new_depth > 0);
      }
    }
    if (i < kIterationsPerKernelCall - 1) {
      __syncthreads();
    }
  }
  
  if (kOutput) {
    depth_map_output(y, x) = depth_shared[shared_mem_index];
  }
}

int InpaintDepthMapWithConvolutionCUDA(
    hipStream_t stream,
    bool use_weighting,
    int max_num_iterations,
    float max_change_rate_threshold,
    float depth_input_scaling_factor,
    hipTextureObject_t gradient_magnitude_div_sqrt2,
    hipTextureObject_t depth_map_input,
    CUDABuffer<uint8_t>* max_change,
    CUDABuffer<float>* depth_map_output,
    CUDABuffer<uint16_t>* block_coordinates,
    uint32_t* pixel_to_inpaint_count) {
  const int width = depth_map_output->width();
  const int height = depth_map_output->height();
  
  const dim3 block_dim(kBlockWidth, kBlockHeight);
  
  const int kBlockOutputSizeX = kBlockWidth - 2 * kIterationsPerKernelCall;
  const int kBlockOutputSizeY = kBlockHeight - 2 * kIterationsPerKernelCall;
  dim3 grid_dim(cuda_util::GetBlockCount(width, kBlockOutputSizeX),
                cuda_util::GetBlockCount(height, kBlockOutputSizeY));
  
  // Initialize variables.
  CHECK_EQ(kBlockWidth, 32);
  CHECK_EQ(kBlockHeight, 32);
  ConvolutionInpaintingInitializeVariablesKernel<32, 32><<<grid_dim, block_dim, 0, stream>>>(
      grid_dim.x, depth_input_scaling_factor, depth_map_input, depth_map_output->ToCUDA(), block_coordinates->ToCUDA());
  CHECK_CUDA_NO_ERROR();
  
  uint16_t* block_activity = new uint16_t[grid_dim.x * grid_dim.y];
  block_coordinates->DownloadPartAsync(0, grid_dim.x * grid_dim.y * sizeof(uint16_t), stream, block_activity);
  hipStreamSynchronize(stream);
  int active_block_count = 0;
  *pixel_to_inpaint_count = 0;
  uint16_t* block_coordinates_cpu = new uint16_t[2 * grid_dim.x * grid_dim.y];
  for (size_t y = 0; y < grid_dim.y; ++ y) {
    for (size_t x = 0; x < grid_dim.x; ++ x) {
      if (block_activity[x + y * grid_dim.x] > 0) {
        block_coordinates_cpu[2 * active_block_count + 0] = x * kBlockOutputSizeX;
        block_coordinates_cpu[2 * active_block_count + 1] = y * kBlockOutputSizeY;
        ++ active_block_count;
        *pixel_to_inpaint_count += block_activity[x + y * grid_dim.x];
      }
    }
  }
  delete[] block_activity;
  if (active_block_count == 0) {
    delete[] block_coordinates_cpu;
    LOG(INFO) << "Depth inpainting converged after iteration: 0";
    return 0;
  }
  block_coordinates->UploadPartAsync(0, 2 * active_block_count * sizeof(uint16_t), stream, block_coordinates_cpu);
  
  uint8_t* max_change_cpu = new uint8_t[grid_dim.x * grid_dim.y];
  
  // Run convolution iterations.
  int i = 0;
  int last_convergence_check_iteration = -9999;
  for (i = 0; i < max_num_iterations; i += kIterationsPerKernelCall) {
    const bool check_convergence = (i - last_convergence_check_iteration >= 25);
    
    dim3 grid_dim_active(active_block_count);
    CHECK_EQ(kBlockWidth, 32);
    CHECK_EQ(kBlockHeight, 32);
    if (use_weighting) {
      if (check_convergence) {
        ConvolutionInpaintingKernelWithWeighting<32, 32, true><<<grid_dim_active, block_dim, 0, stream>>>(
            block_coordinates->ToCUDA(),
            depth_map_input,
            gradient_magnitude_div_sqrt2,
            max_change->ToCUDA(),
            max_change_rate_threshold,
            depth_map_output->ToCUDA());
      } else {
        ConvolutionInpaintingKernelWithWeighting<32, 32, false><<<grid_dim_active, block_dim, 0, stream>>>(
            block_coordinates->ToCUDA(),
            depth_map_input,
            gradient_magnitude_div_sqrt2,
            max_change->ToCUDA(),
            max_change_rate_threshold,
            depth_map_output->ToCUDA());
      }
    } else {
      if (check_convergence) {
        ConvolutionInpaintingKernel<32, 32, true><<<grid_dim_active, block_dim, 0, stream>>>(
            block_coordinates->ToCUDA(),
            depth_map_input,
            max_change->ToCUDA(),
            max_change_rate_threshold,
            depth_map_output->ToCUDA());
      } else {
        ConvolutionInpaintingKernel<32, 32, false><<<grid_dim_active, block_dim, 0, stream>>>(
            block_coordinates->ToCUDA(),
            depth_map_input,
            max_change->ToCUDA(),
            max_change_rate_threshold,
            depth_map_output->ToCUDA());
      }
    }
    
    if (check_convergence) {
      max_change->DownloadPartAsync(0, active_block_count * sizeof(uint8_t), stream, max_change_cpu);
      hipStreamSynchronize(stream);
      int new_active_block_count = 0;
      for (int j = 0, end = active_block_count; j < end; j ++) {
        if (max_change_cpu[j]) {
          ++ new_active_block_count;
        }
      }
      if (new_active_block_count == 0) {
        i += kIterationsPerKernelCall;  // For correct iteration count logging.
        break;
      }
      last_convergence_check_iteration = i;
    }
  }
  
  delete[] max_change_cpu;
  delete[] block_coordinates_cpu;
  CHECK_CUDA_NO_ERROR();
  
  if (i < max_num_iterations) {
    LOG(INFO) << "Depth inpainting converged after iteration: " << i;
  } else {
    LOG(WARNING) << "Depth inpainting used maximum iteration count: " << i;
  }
  return i;
}

}
